#include "hip/hip_runtime.h"
/*
 *  remote_spike.cu
 *
 *  This file is part of NEST GPU.
 *
 *  Copyright (C) 2021 The NEST Initiative
 *
 *  NEST GPU is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  NEST GPU is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with NEST GPU.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

__constant__ bool have_remote_spike_height;

#include <config.h>

#include <stdio.h>
#include <stdlib.h>

#include "cuda_error.h"
#include "utilities.h"
#include "spike_buffer.h"
#include "getRealTime.h"

#include "remote_spike.h"

#include "scan.h"
#include "utilities.h"
#include "remote_connect.h"


// Simple kernel for pushing remote spikes in local spike buffers
// Version without spike multiplicity array (spike_height) 
__global__ void PushSpikeFromRemote(int n_spikes, int *spike_buffer_id)
{
  int i_spike = threadIdx.x + blockIdx.x * blockDim.x;
  if (i_spike<n_spikes) {
    int isb = spike_buffer_id[i_spike];
    PushSpike(isb, 1.0);
  }
}

__device__ int NExternalTargetHost;
__device__ int MaxSpikePerHost;

int *d_ExternalSpikeNum;
__device__ int *ExternalSpikeNum;

int *d_ExternalSpikeSourceNode; // [MaxSpikeNum];
__device__ int *ExternalSpikeSourceNode;

float *d_ExternalSpikeHeight; // [MaxSpikeNum];
__device__ float *ExternalSpikeHeight;

int *d_ExternalTargetSpikeNum;
__device__ int *ExternalTargetSpikeNum;

int *d_ExternalTargetSpikeNodeId;
__device__ int *ExternalTargetSpikeNodeId;

float *d_ExternalTargetSpikeHeight;
__device__ float *ExternalTargetSpikeHeight;

//int *d_NExternalNodeTargetHost;
__device__ int *NExternalNodeTargetHost;

//int **d_ExternalNodeTargetHostId;
__device__ int **ExternalNodeTargetHostId;

//int **d_ExternalNodeId;
__device__ int **ExternalNodeId;

//int *d_ExternalSourceSpikeNum;
//__device__ int *ExternalSourceSpikeNum;

int *d_ExternalSourceSpikeNodeId;
__device__ int *ExternalSourceSpikeNodeId;

float *d_ExternalSourceSpikeHeight;
__device__ float *ExternalSourceSpikeHeight;

int *d_ExternalTargetSpikeIdx0;
__device__ int *ExternalTargetSpikeIdx0;
int *h_ExternalTargetSpikeIdx0;

int *d_ExternalSourceSpikeIdx0;

int *h_ExternalTargetSpikeNum;
int *h_ExternalSourceSpikeNum;
int *h_ExternalSourceSpikeIdx0;
int *h_ExternalTargetSpikeNodeId;
int *h_ExternalSourceSpikeNodeId;

//int *h_ExternalSpikeNodeId;

float *h_ExternalSpikeHeight;

// Push in a dedicated array the spikes that must be sent externally
__device__ void PushExternalSpike(int i_source, float height)
{
  int pos = atomicAdd(ExternalSpikeNum, 1);
  if (pos>=MaxSpikePerHost) {
    printf("Number of spikes larger than MaxSpikePerHost: %d\n", MaxSpikePerHost);
    *ExternalSpikeNum = MaxSpikePerHost;
    return;
  }
  ExternalSpikeSourceNode[pos] = i_source;
  ExternalSpikeHeight[pos] = height;
}

// Push in a dedicated array the spikes that must be sent externally
// (version without spike height)
__device__ void PushExternalSpike(int i_source)
{
  int pos = atomicAdd(ExternalSpikeNum, 1);
  if (pos>=MaxSpikePerHost) {
    printf("Number of spikes larger than MaxSpikePerHost: %d\n", MaxSpikePerHost);
    *ExternalSpikeNum = MaxSpikePerHost;
    return;
  }
  ExternalSpikeSourceNode[pos] = i_source;
}

// Count the spikes that must be sent externally for each target host
__global__ void countExternalSpikesPerTargetHost()
{
  const int i_spike = blockIdx.x;
  if (i_spike < *ExternalSpikeNum) {
    //printf("ExternalSpikeNum: %d\ti_spike: %d\n", *ExternalSpikeNum, i_spike);
    int i_source = ExternalSpikeSourceNode[i_spike];
    //printf("i_source: %d\n", i_source);
    int Nth = NExternalNodeTargetHost[i_source];
    //printf("Nth: %d\n", Nth);
    
    for (int ith=threadIdx.x; ith<Nth; ith+=blockDim.x){
      //printf("ith: %d\n", ith);
      int target_host_id = ExternalNodeTargetHostId[i_source][ith];
      //printf("target_host_id: %d\n", target_host_id);
      //int remote_node_id = ExternalNodeId[i_source][ith];
      //printf("remote_node_id: %d\n", remote_node_id);
      //int pos =
      atomicAdd(&ExternalTargetSpikeNum[target_host_id], 1);
      //printf("pos: %d\n", pos);
    }
  }
}

// Organize the spikes that must be sent externally for each target host
__global__ void organizeExternalSpikesPerTargetHost()
{
  const int i_spike = blockIdx.x;
  if (i_spike < *ExternalSpikeNum) {
    //printf("ExternalSpikeNum: %d\ti_spike: %d\n", *ExternalSpikeNum, i_spike);
    int i_source = ExternalSpikeSourceNode[i_spike];
    //printf("i_source: %d\n", i_source);
    int Nth = NExternalNodeTargetHost[i_source];
    //printf("Nth: %d\n", Nth);
    
    for (int ith=threadIdx.x; ith<Nth; ith+=blockDim.x){
      //printf("ith: %d\n", ith);
      int target_host_id = ExternalNodeTargetHostId[i_source][ith];
      //printf("target_host_id: %d\n", target_host_id);
      int remote_node_id = ExternalNodeId[i_source][ith];
      //printf("remote_node_id: %d\n", remote_node_id);
      int pos = atomicAdd(&ExternalTargetSpikeNum[target_host_id], 1);
      //printf("pos: %d\n", pos);
      int i_arr = ExternalTargetSpikeIdx0[target_host_id] + pos;
      ExternalTargetSpikeNodeId[i_arr] = remote_node_id;
      if (have_remote_spike_height) {
	float height = ExternalSpikeHeight[i_spike];
	//printf("height: %f\n", height);
	ExternalTargetSpikeHeight[i_arr] = height;
	//printf("ExternalTargetSpikeHeight assigned\n");
      }
    }
  }
}

// reset external spike counters
int NESTGPU::ExternalSpikeReset()
{
  gpuErrchk(hipMemset(d_ExternalSpikeNum, 0, sizeof(int)));
  gpuErrchk(hipMemset(d_ExternalTargetSpikeNum, 0, n_hosts_*sizeof(int)));
  
  return 0;
}

// initialize external spike arrays
int NESTGPU::ExternalSpikeInit()
{
  SendSpikeToRemote_comm_time_ = 0;
  RecvSpikeFromRemote_comm_time_ = 0;
  
  SendSpikeToRemote_CUDAcp_time_ = 0;
  RecvSpikeFromRemote_CUDAcp_time_ = 0;

  //int *h_NExternalNodeTargetHost = new int[n_node];
  //int **h_ExternalNodeTargetHostId = new int*[n_node];
  //int **h_ExternalNodeId = new int*[n_node];

  h_ExternalTargetSpikeIdx0 = new int[n_hosts_+1];
  //h_ExternalSpikeNodeId = new int[max_spike_per_host_];
  h_ExternalTargetSpikeNum = new int [n_hosts_];
  h_ExternalSourceSpikeNum = new int[n_hosts_];
  h_ExternalSourceSpikeIdx0 = new int[n_hosts_ + 1];
  h_ExternalTargetSpikeNodeId = new int[max_remote_spike_num_];
  h_ExternalSourceSpikeNodeId = new int[max_remote_spike_num_];

  CUDAMALLOCCTRL("&d_ExternalSpikeNum",&d_ExternalSpikeNum, sizeof(int));
  CUDAMALLOCCTRL("&d_ExternalSpikeSourceNode",&d_ExternalSpikeSourceNode,
		       max_spike_per_host_*sizeof(int));
  
  if (remote_spike_height_) {
    h_ExternalSpikeHeight = new float[max_spike_per_host_];
    CUDAMALLOCCTRL("&d_ExternalSpikeHeight",&d_ExternalSpikeHeight,
		   max_spike_per_host_*sizeof(float));
    CUDAMALLOCCTRL("&d_ExternalTargetSpikeHeight",&d_ExternalTargetSpikeHeight,
		   max_remote_spike_num_*sizeof(float));
    CUDAMALLOCCTRL("&d_ExternalSourceSpikeHeight",&d_ExternalSourceSpikeHeight,
		   max_remote_spike_num_*sizeof(float));
  }
  
  CUDAMALLOCCTRL("&d_ExternalTargetSpikeNum",&d_ExternalTargetSpikeNum,
		 n_hosts_*sizeof(int));

  //printf("n_hosts, max_spike_per_host: %d %d\n", n_hosts, max_spike_per_host);

  CUDAMALLOCCTRL("&d_ExternalTargetSpikeNodeId",
		 &d_ExternalTargetSpikeNodeId,
		 max_remote_spike_num_*sizeof(int));
  
  //CUDAMALLOCCTRL("&d_ExternalSourceSpikeNum",&d_ExternalSourceSpikeNum,
  //n_hosts*sizeof(int));
  CUDAMALLOCCTRL("&d_ExternalSourceSpikeNodeId",&d_ExternalSourceSpikeNodeId,
		 max_remote_spike_num_*sizeof(int));
  CUDAMALLOCCTRL("&d_ExternalTargetSpikeIdx0",&d_ExternalTargetSpikeIdx0,
		 (n_hosts_ + 1)*sizeof(int));

  CUDAMALLOCCTRL("&d_ExternalSourceSpikeIdx0",&d_ExternalSourceSpikeIdx0,
		 (n_hosts_ + 1)*sizeof(int));
  
  //CUDAMALLOCCTRL("&d_NExternalNodeTargetHost",&d_NExternalNodeTargetHost,
  //n_node*sizeof(int));
  //CUDAMALLOCCTRL("&d_ExternalNodeTargetHostId",&d_ExternalNodeTargetHostId,
  //n_node*sizeof(int*));
  //CUDAMALLOCCTRL("&d_ExternalNodeId",&d_ExternalNodeId, n_node*sizeof(int*));

  if (remote_spike_height_) {
    DeviceExternalSpikeInit<<<1,1>>>(n_hosts_, max_spike_per_host_,
				     d_ExternalSpikeNum,
				     d_ExternalSpikeSourceNode,
				     d_ExternalSpikeHeight,
				     d_ExternalTargetSpikeNum,
				     d_ExternalTargetSpikeIdx0,
				     d_ExternalTargetSpikeNodeId,
				     d_ExternalTargetSpikeHeight,
				     d_n_target_hosts,
				     d_node_target_hosts,
				     d_node_target_host_i_map
				     );
  }
  else {
    DeviceExternalSpikeInit<<<1,1>>>(n_hosts_, max_spike_per_host_,
				     d_ExternalSpikeNum,
				     d_ExternalSpikeSourceNode,
				     d_ExternalTargetSpikeNum,
				     d_ExternalTargetSpikeIdx0,
				     d_ExternalTargetSpikeNodeId,
				     d_n_target_hosts,
				     d_node_target_hosts,
				     d_node_target_host_i_map
				     );  
  }
  //delete[] h_NExternalNodeTargetHost;
  //delete[] h_ExternalNodeTargetHostId;
  //delete[] h_ExternalNodeId;

  return 0;
}

// initialize external spike array pointers in the GPU
__global__ void DeviceExternalSpikeInit(int n_hosts,
					int max_spike_per_host,
					int *ext_spike_num,
					int *ext_spike_source_node,
					float *ext_spike_height,
					int *ext_target_spike_num,
					int *ext_target_spike_idx0,
					int *ext_target_spike_node_id,
					float *ext_target_spike_height,
					int *n_ext_node_target_host,
					int **ext_node_target_host_id,
					int **ext_node_id
					)
  
{
  NExternalTargetHost = n_hosts;
  MaxSpikePerHost =  max_spike_per_host;
  ExternalSpikeNum = ext_spike_num;
  ExternalSpikeSourceNode = ext_spike_source_node;
  ExternalSpikeHeight = ext_spike_height;
  ExternalTargetSpikeNum = ext_target_spike_num;
  ExternalTargetSpikeIdx0 = ext_target_spike_idx0,
  ExternalTargetSpikeNodeId = ext_target_spike_node_id;
  ExternalTargetSpikeHeight = ext_target_spike_height;
  NExternalNodeTargetHost = n_ext_node_target_host;
  ExternalNodeTargetHostId = ext_node_target_host_id;
  ExternalNodeId = ext_node_id;
  *ExternalSpikeNum = 0;
  for (int ith=0; ith<NExternalTargetHost; ith++) {
    ExternalTargetSpikeNum[ith] = 0;
  }  
}

// initialize external spike array pointers in the GPU
// (version without spike height)
__global__ void DeviceExternalSpikeInit(int n_hosts,
					int max_spike_per_host,
					int *ext_spike_num,
					int *ext_spike_source_node,
					int *ext_target_spike_num,
					int *ext_target_spike_idx0,
					int *ext_target_spike_node_id,
					int *n_ext_node_target_host,
					int **ext_node_target_host_id,
					int **ext_node_id
					)
  
{
  NExternalTargetHost = n_hosts;
  MaxSpikePerHost =  max_spike_per_host;
  ExternalSpikeNum = ext_spike_num;
  ExternalSpikeSourceNode = ext_spike_source_node;
  ExternalSpikeHeight = NULL;
  ExternalTargetSpikeNum = ext_target_spike_num;
  ExternalTargetSpikeIdx0 = ext_target_spike_idx0,
  ExternalTargetSpikeNodeId = ext_target_spike_node_id;
  ExternalTargetSpikeHeight = NULL;
  NExternalNodeTargetHost = n_ext_node_target_host;
  ExternalNodeTargetHostId = ext_node_target_host_id;
  ExternalNodeId = ext_node_id;
  *ExternalSpikeNum = 0;
  for (int ith=0; ith<NExternalTargetHost; ith++) {
    ExternalTargetSpikeNum[ith] = 0;
  }  
}

int NESTGPU::organizeExternalSpikes(int n_ext_spikes)
{
  countExternalSpikesPerTargetHost<<<n_ext_spikes, 1024>>>();
  CUDASYNC;
  prefix_scan(d_ExternalTargetSpikeIdx0, d_ExternalTargetSpikeNum,
	      n_hosts_+1, true);
  DBGCUDASYNC;
  gpuErrchk(hipMemset(d_ExternalTargetSpikeNum, 0, n_hosts_*sizeof(int)));
  organizeExternalSpikesPerTargetHost<<<n_ext_spikes, 1024>>>();
  CUDASYNC;
  
  return 0;  
}

// pack spikes received from remote hosts
// and copy them to GPU memory
int NESTGPU::CopySpikeFromRemote()
{
  int n_spike_tot = 0;
  h_ExternalSourceSpikeIdx0[0] = 0;
  // loop on hosts
  for (int i_host=0; i_host<n_hosts_; i_host++) {
    int n_spike = h_ExternalSourceSpikeNum[i_host];
    h_ExternalSourceSpikeIdx0[i_host+1] =
      h_ExternalSourceSpikeIdx0[i_host] + n_spike;
    for (int i_spike=0; i_spike<n_spike; i_spike++) {
      // pack spikes received from remote hosts
      h_ExternalSourceSpikeNodeId[n_spike_tot] =
	h_ExternalSourceSpikeNodeId[i_host*max_spike_per_host_ + i_spike];
      n_spike_tot++;
    }
  }
  
  if (n_spike_tot >= max_remote_spike_num_) {
    throw ngpu_exception
      (std::string("Number of spikes to be received remotely ")
       + std::to_string(n_spike_tot)
       + " larger than limit " + std::to_string(max_remote_spike_num_));
  }
  
  if (n_spike_tot>0) {
    double time_mark = getRealTime();
    // Memcopy will be synchronized
    // copy to GPU memory cumulative sum of number of spikes per source host
    gpuErrchk(hipMemcpyAsync(d_ExternalSourceSpikeIdx0,
			      h_ExternalSourceSpikeIdx0,
			      (n_hosts_+1)*sizeof(int), hipMemcpyHostToDevice));
    DBGCUDASYNC;
    // copy to GPU memory packed spikes from remote hosts
    gpuErrchk(hipMemcpyAsync(d_ExternalSourceSpikeNodeId,
			      h_ExternalSourceSpikeNodeId,
			      n_spike_tot*sizeof(int), hipMemcpyHostToDevice));
    DBGCUDASYNC;
    RecvSpikeFromRemote_CUDAcp_time_ += (getRealTime() - time_mark);
    // convert node map indexes to spike buffer indexes
    MapIndexToSpikeBufferKernel<<<n_hosts_, 1024>>>(n_hosts_,
						   d_ExternalSourceSpikeIdx0,
						   d_ExternalSourceSpikeNodeId);
    DBGCUDASYNC;
    // convert node group indexes to spike buffer indexes
    // by adding the index of the first node of the node group  
    //AddOffset<<<(n_spike_tot+1023)/1024, 1024>>>
    //  (n_spike_tot, d_ExternalSourceSpikeNodeId, i_remote_node_0);
    //gpuErrchk( hipPeekAtLastError() );
    //hipDeviceSynchronize();
    // push remote spikes in local spike buffers
    PushSpikeFromRemote<<<(n_spike_tot+1023)/1024, 1024>>>
      (n_spike_tot, d_ExternalSourceSpikeNodeId);
    DBGCUDASYNC;
  }
  
  return n_spike_tot;
}

